#include "hip/hip_runtime.h"
/********************************************************************
*  sample.cu
*  This is a example of the CUDA program.
*********************************************************************/

#include <windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <iostream>
#include <map>
#include <vector>
#include <list>

#include <irrlicht.h>
#pragma comment (lib, "Irrlicht.lib")

using namespace irr;
using namespace core;
using namespace scene;
using namespace video;
using namespace io;
using namespace gui;

struct MeshData
{
	// Irrlicht�p
	int			indexCount;
	int			irr_vertexCount;
	u16         *irr_indices;
	S3DVertex   *irr_vertices;
	// CUDA�p
	float		*cuda_vertices;
	int			cuda_vertexCount;
	dim3		dimGrid;
	float		*device_result;
	// Irrlicht��CUDA�̊֘A�t��
	std::map<int, int> irr_cuda_map;
};

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

#endif
/************************************************************************/
/* Example                                                              */
/************************************************************************/
__global__ static void HelloCUDA(float* vertex, int vertexCount, float(dt))
{
	int index = blockIdx.x * 256 + threadIdx.y * 16 + threadIdx.x;

	if (index >= vertexCount) 
	{ 
		return;
	}

	// �K���ɒ��_�A�j���[�V����
	vertex[3*index+0] += 0.002f*cos(2.0f*dt+0.1f*(float)threadIdx.x);
	vertex[3*index+1] += 0.001f*sin(dt);
	vertex[3*index+2] += 0.002f*cos(3.0f*dt+0.1f*(float)threadIdx.y);
}


/************************************************************************/
/* Irrlicht                                                             */
/************************************************************************/

/************************************************************************/
/* HelloCUDA                                                            */
/************************************************************************/
int main(int argc, char* argv[])
{
	// �R�������Ă��Ȃ��ƃt�@�C�������E�E�E
	std::locale::global(std::locale("japanese"));

	// CUDA�̏�����
	if(!InitCUDA()) {
		return 0;
	}

	// window size
	const core::dimension2d<u32>& windowSize = (core::dimension2d<u32>(640, 480));

	// �f�o�C�X�̍쐬
	IrrlichtDevice *device = createDevice( EDT_OPENGL, windowSize, 32,
		false, true, false, 0);

	if (!device)
	{
		return false;
	}
	IVideoDriver* driver = device->getVideoDriver();
	ISceneManager *smgr = device->getSceneManager();

	// �J�����̐ݒ�
	ICameraSceneNode *camera = smgr->addCameraSceneNodeMaya(0);

	// ���b�V���f�[�^���t�@�C������ǂݍ��݁iIrrlicht�p�j
	IAnimatedMesh *cubeMesh = smgr->getMesh("../../media/moko_one.3ds");
	IAnimatedMeshSceneNode *cubeNode = smgr->addAnimatedMeshSceneNode(cubeMesh, 0, -1, vector3df(0, 80, 0), vector3df(-90,0,0), vector3df(1,1,1), false);
	cubeNode->setMaterialFlag(video::EMF_LIGHTING, false);
	cubeNode->setMaterialFlag(video::EMF_BACK_FACE_CULLING, true);

	// �n�ʂ̍쐬�iIrrlicht�p�j
	IAnimatedMesh *planemesh = smgr->addHillPlaneMesh("myHill", dimension2d<f32>(24, 24), dimension2d<u32>(100, 100));
	ISceneNode *q3sn = smgr->addOctTreeSceneNode(planemesh);
	q3sn->setMaterialFlag(video::EMF_LIGHTING, false);
	q3sn->setMaterialTexture(0, driver->getTexture("../../media/wall.jpg"));

	// create skybox
	driver->setTextureCreationFlag(video::ETCF_CREATE_MIP_MAPS, false);
	smgr->addSkyBoxSceneNode(
		driver->getTexture("../../media/irrlicht2_up.jpg"),
		driver->getTexture("../../media/irrlicht2_dn.jpg"),
		driver->getTexture("../../media/irrlicht2_lf.jpg"),
		driver->getTexture("../../media/irrlicht2_rt.jpg"),
		driver->getTexture("../../media/irrlicht2_ft.jpg"),
		driver->getTexture("../../media/irrlicht2_bk.jpg"));

	driver->setTextureCreationFlag(video::ETCF_CREATE_MIP_MAPS, true);

	int cMeshBuffer, cMeshBufferCount, j;
	// MeshBuffer
	IMeshBuffer **mb;
	// MeshData struct
	MeshData *testMesh;

	std::cout << "get Mesh Buffer : ";
	cMeshBufferCount = cubeMesh->getMesh(0)->getMeshBufferCount();
	std::cout << cMeshBufferCount << std::endl;
	mb = new IMeshBuffer*[cMeshBufferCount];
	testMesh = new MeshData[cMeshBufferCount];

	// getMesh�Ŏ擾�������b�V�����̎擾
	for (cMeshBuffer=0; cMeshBuffer<cMeshBufferCount; cMeshBuffer++) 
	{
		// ���b�V���o�b�t�@�̎擾
		mb[cMeshBuffer] = cubeMesh->getMesh(0)->getMeshBuffer(cMeshBuffer); 

		// ���_���̃|�C���^���擾�B���̒l������������Β��_���W��ύX�ł���B
		testMesh[cMeshBuffer].irr_vertices = (irr::video::S3DVertex*)mb[cMeshBuffer]->getVertices();
		testMesh[cMeshBuffer].irr_vertexCount = mb[cMeshBuffer]->getVertexCount();
		// �ʂ̃C���f�b�N�X�̃|�C���^���擾�B���̒l������������Ζʂ̃C���f�b�N�X��ύX�ł���B
		testMesh[cMeshBuffer].irr_indices  = mb[cMeshBuffer]->getIndices(); 
		testMesh[cMeshBuffer].indexCount = mb[cMeshBuffer]->getIndexCount();
		// ���̑�������
		testMesh[cMeshBuffer].irr_cuda_map.clear();
		testMesh[cMeshBuffer].cuda_vertices = NULL;
		testMesh[cMeshBuffer].device_result = NULL;
		std::cout << mb[cMeshBuffer]->getVertexCount() << ":" << mb[cMeshBuffer]->getIndexCount() << std::endl;
	} 

	// �ǂݍ��񂾃��b�V���f�[�^�̍œK��
	// 3ds�t�@�C���i.obj���H�j��Irrlicht�œǂݍ��ނƁA�iIndex����Vertex���j�ƂȂ�AIndex����ӂŁAVertex���d������B
	// �������_�����x��CUDA�Ōv�Z�������Ȃ��̂ŁAVertex�̏d�����Ȃ����B
	// ���̂��߂ɂ܂��A�d���̂Ȃ�Index�Əd���̂���Index�̃}�b�v���쐬
	std::map<int, int> index_map;
	std::map<int, int> index2_map;
	std::map<int, S3DVertex> vertex_map;
	int count = 0;

	for (cMeshBuffer = 0; cMeshBuffer < cMeshBufferCount; cMeshBuffer++)
	{
		// Vertex����Index���ł͂Ȃ��ꍇ�͏������Ȃ�
		if (mb[cMeshBuffer]->getVertexCount() != mb[cMeshBuffer]->getIndexCount())
		{
			continue;
		}
		count = 0;
		index_map.clear();
		index2_map.clear();
		vertex_map.clear();
		// �i�d�����Ă���jVertex�̐���Index�̐������J��Ԃ�
		for (int i=0; i<testMesh[cMeshBuffer].irr_vertexCount; i++)
		{
			// i�Ԗڂ̃C���f�b�N�X�ƒ��_���W���擾
			int iIndex = testMesh[cMeshBuffer].irr_indices[i];
			vector3df iVertex = testMesh[cMeshBuffer].irr_vertices[iIndex].Pos;
			// �d���������_���ǂ����̊m�F
			bool isFirst = true;
			// 0����i-1�Ԗڂ̃C���f�b�N�X�̒��_���W�Ɣ�r
			for (int j=0; j<i; j++)
			{
				// j�Ԗڂ̃C���f�b�N�X�ƒ��_���W���擾
				int jIndex = testMesh[cMeshBuffer].irr_indices[j];
				vector3df jVertex = testMesh[cMeshBuffer].irr_vertices[jIndex].Pos;
				// �����C���f�b�N�X�ԍ��̏ꍇ
				if (iVertex == jVertex)
				{
					// i�Ԗڂ̃C���f�b�N�X�́Aj�Ԗڂ̃C���f�b�N�X�Ɠ����Ƃ�����w���Ă���
					index_map.insert(std::make_pair(i, j));
					// Irrlicht->CUDA��Index�̃}�b�s���O
					int cuda_index = index2_map.find(j)->second;
					testMesh[cMeshBuffer].irr_cuda_map.insert(std::make_pair(i, cuda_index));
					isFirst = false;
					break;
				}
			}
			// �d�����钸�_���Ȃ��ꍇ��Index�ƒ��_���֘A�t��
			if (isFirst)
			{
				// Irrlicht��Index���d���̂���Index
				index_map.insert(std::make_pair(i, i));
				// �ƂтƂт�Index���A����Index
				index2_map.insert(std::make_pair(i, count));
				// Irrlicht->CUDA��Index�̃}�b�s���O
				testMesh[cMeshBuffer].irr_cuda_map.insert(std::make_pair(i, count));
				// �A��Index�ƒ��_���̃}�b�v
				vertex_map.insert(std::make_pair(count, testMesh[cMeshBuffer].irr_vertices[iIndex]));
				count++;
			}
		}

		// index-vertex data for cuda
		testMesh[cMeshBuffer].cuda_vertexCount = vertex_map.size();
		// CUDA���ŃO���[�o�����������m��
		cutilSafeCall( hipMalloc((void**) &testMesh[cMeshBuffer].device_result, 
			sizeof(float) * testMesh[cMeshBuffer].cuda_vertexCount*3));
		// CUDA�ɓn���̂�Host�Ƃ��Ċm��
		cutilSafeCall( hipHostMalloc((void**) &testMesh[cMeshBuffer].cuda_vertices, 
			sizeof(float) * testMesh[cMeshBuffer].cuda_vertexCount*3));

		// ���b�V������|���S�������擾���z��Ɋi�[
		for (j=0; j<testMesh[cMeshBuffer].cuda_vertexCount; j++)
		{
			testMesh[cMeshBuffer].cuda_vertices[3*j]   =  vertex_map[j].Pos.X;
			testMesh[cMeshBuffer].cuda_vertices[3*j+1] =  vertex_map[j].Pos.Y;
			testMesh[cMeshBuffer].cuda_vertices[3*j+2] =  vertex_map[j].Pos.Z;
		}

		// �������_�f�[�^��]��
		cutilSafeCall( hipMemcpy(testMesh[cMeshBuffer].device_result, testMesh[cMeshBuffer].cuda_vertices, 
			sizeof(float) * testMesh[cMeshBuffer].cuda_vertexCount*3, hipMemcpyHostToDevice));
	}

	// CUDA�Ŋe���_���v�Z���邽�߂Ƀu���b�N��������
	int lastVertexCount = 0, blockNum = 0;
	// 1�u���b�N��16x16��256�X���b�h�ɌŒ�
	dim3 dimBlock(16, 16, 1);

	for (cMeshBuffer=0; cMeshBuffer<cMeshBufferCount; cMeshBuffer++) 
	{
		// ��ԍŌ�̃u���b�N���������钸�_��
		lastVertexCount = testMesh[cMeshBuffer].cuda_vertexCount%256;
		// 0�̏ꍇ�A����؂��̂Ńu���b�N���͒��_��/256
		if (lastVertexCount == 0)
		{
			blockNum = testMesh[cMeshBuffer].cuda_vertexCount/256;
		}
		// 0���傫���ꍇ�A����؂�Ȃ��̂Ŏc�������_����������u���b�N���K�v
		else
		{
			blockNum = testMesh[cMeshBuffer].cuda_vertexCount/256+1;
		}
		std::cout << "blockNum=" << blockNum << std::endl;
		// 1�O���b�h��blockNum�u���b�N�Ɍ���
		testMesh[cMeshBuffer].dimGrid = dim3(blockNum, 1, 1);
	}

	// FPS�v�Z�p
	wchar_t tmp[32] = L"";
	int fps, lastFPS  = -1;

	// ���_�A�j���[�V�����p
	static float dt = 0.0f;

	while(device->run())
	{
		for (cMeshBuffer=0; cMeshBuffer<cMeshBufferCount; cMeshBuffer++) 
		{
			// CUDA�Œ��_�A�j���[�V����
			HelloCUDA<<<testMesh[cMeshBuffer].dimGrid, dimBlock>>>(testMesh[cMeshBuffer].device_result, testMesh[cMeshBuffer].cuda_vertexCount, dt);
			cutilCheckMsg("Kernel execution failed\n");

			// �v�Z���ʂ̓]��
			cutilSafeCall( hipMemcpy(testMesh[cMeshBuffer].cuda_vertices, testMesh[cMeshBuffer].device_result, 
				sizeof(float) * testMesh[cMeshBuffer].cuda_vertexCount * 3, hipMemcpyDeviceToHost));

			// �v�Z���ʂ��璸�_���W���X�V
			for (int i=0; i<testMesh[cMeshBuffer].indexCount; i++)
			{
				// Irrlict�̒��_�ԍ���CUDA�̒��_�ԍ�
				int irr_index = testMesh[cMeshBuffer].irr_indices[i];
				int cuda_index = testMesh[cMeshBuffer].irr_cuda_map.find(i)->second;
				// Irrlicht����mb_vertices[i]��OK
				testMesh[cMeshBuffer].irr_vertices[irr_index].Pos.X = testMesh[cMeshBuffer].cuda_vertices[3*cuda_index+0];
				testMesh[cMeshBuffer].irr_vertices[irr_index].Pos.Y = testMesh[cMeshBuffer].cuda_vertices[3*cuda_index+1];
				testMesh[cMeshBuffer].irr_vertices[irr_index].Pos.Z = testMesh[cMeshBuffer].cuda_vertices[3*cuda_index+2];
			}
		} 
		dt += 0.001f;
		if (GetAsyncKeyState(VK_SPACE))
		{

		}
		else if (GetAsyncKeyState(VK_ESCAPE)) 
		{
			break;
		}

		// �`�揈��
		driver->beginScene(true, true, SColor(0,200,200,200));
		smgr->drawAll();
		driver->endScene();

		//FPS�擾/�\��
		fps = driver->getFPS();
		if(lastFPS != fps)
		{
			swprintf(tmp, 32, L"Irrlicht fps:%d", fps);
			device->setWindowCaption(tmp);
			lastFPS = fps;
		}
	}
	// Irrlicht�̏I��
	device->drop();

	// �������̊J��
	for (cMeshBuffer=0; cMeshBuffer<cMeshBufferCount; cMeshBuffer++)
	{
		if (testMesh[cMeshBuffer].cuda_vertices != NULL)
		{
			cutilSafeCall( hipHostFree(testMesh[cMeshBuffer].cuda_vertices) );
		}
		if (testMesh[cMeshBuffer].device_result != NULL)
		{
			cutilSafeCall( hipFree(testMesh[cMeshBuffer].device_result));
		}
	}
	delete [] testMesh;
	delete [] mb;

	return 0;
}
